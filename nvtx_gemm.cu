#include "hip/hip_runtime.h"
// naive_gemm.cu  ── plain row-major C = A × B (O(N3))
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <nvToolsExt.h>
#include <hip/hip_runtime_api.h>


__global__ void matmul_naive(const float* A, const float* B,
                             float* C, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.f;
        for (int k = 0; k < N; ++k)
            sum += A[row * N + k] * B[k * N + col];
        C[row * N + col] = sum;
    }
}

int main(int argc, char** argv)
{
    const int N = (argc > 1) ? std::stoi(argv[1]) : 1024;
    const size_t bytes = size_t(N) * N * sizeof(float);

    // float *hA = new float[N * N], *hB = new float[N * N], *hC = new float[N * N]; 
    float *hA, *hB, *hC;
    hipHostMalloc((void**)&hA, bytes);
    hipHostMalloc((void**)&hB, bytes);
    hipHostMalloc((void**)&hC, bytes);

    for (int i = 0; i < N * N; ++i) hA[i] = hB[i] = 1.f;

    unsigned long long int block = 512;
    unsigned long long int grid = (N + block - 1) / block;
    
    float *dA, *dB, *dC;
    hipMalloc(&dA, bytes);
    hipMalloc(&dB, bytes);
    hipMalloc(&dC, bytes);

    auto start = std::chrono::high_resolution_clock::now();
    hipMemcpy(dA, hA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, bytes, hipMemcpyHostToDevice);

    hipProfilerStart();
    matmul_naive<<<grid, block>>>(dA, dB, dC, N);
    hipProfilerStop();

    hipMemcpy(hC, dC, bytes, hipMemcpyDeviceToHost);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Elapsed time: " << elapsed.count() << " seconds\n";

    std::cout << "C[0] = " << hC[0] << '\n';         // quick check (== N)
    hipFree(dA); hipFree(dB); hipFree(dC);
    // delete[] hA; delete[] hB; delete[] hC;
    return 0;
}
