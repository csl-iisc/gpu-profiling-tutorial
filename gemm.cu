// naive_gemm.cu  ── plain row-major C = A × B (O(N3))
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void matmul_naive(const float* A, const float* B,
                             float* C, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.f;
        for (int k = 0; k < N; ++k)
            sum += A[row * N + k] * B[k * N + col];
        C[row * N + col] = sum;
    }
}

int main(int argc, char** argv)
{
    const int N = (argc > 1) ? std::stoi(argv[1]) : 1024;
    const size_t bytes = size_t(N) * N * sizeof(float);

    float *hA = new float[N * N], *hB = new float[N * N], *hC = new float[N * N]; 

    for (int i = 0; i < N * N; ++i) hA[i] = hB[i] = 1.f;

    float *dA, *dB, *dC;
    hipMalloc(&dA, bytes);
    hipMalloc(&dB, bytes);
    hipMalloc(&dC, bytes);

    dim3 block(32, 32);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    auto start = std::chrono::high_resolution_clock::now();
    hipMemcpy(dA, hA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, bytes, hipMemcpyHostToDevice);

    matmul_naive<<<grid, block>>>(dA, dB, dC, N);

    hipMemcpy(hC, dC, bytes, hipMemcpyDeviceToHost);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Elapsed time: " << elapsed.count() << " seconds\n";

    std::cout << "C[0] = " << hC[0] << '\n';         // quick check (== N)
    hipFree(dA); hipFree(dB); hipFree(dC);
    delete[] hA; delete[] hB; delete[] hC;
    return 0;
}
